#include "hip/hip_runtime.h"
#include "GEMMBatchSolver.h"
#include <hipblas.h>
#include <thrust/device_ptr.h>
#include <thrust/gather.h>
#include <thrust/for_each.h>
#include <thrust/execution_policy.h>

GEMMBatchSolver::GEMMBatchSolver()
    : m_handle(nullptr)
{
    hipblasCreate(&m_handle);
}

GEMMBatchSolver::~GEMMBatchSolver()
{
    std::cout << "Destroying GEMMBatchSolver..." << std::endl;
  //  hipStreamSynchronize(m_stream); // Ensure all operations on the stream are completed before cleanup
    hipblasDestroy(m_handle);
}

void GEMMBatchSolver::setStream(hipStream_t stream)
{
    m_stream = stream;
    hipblasSetStream(m_handle, m_stream);
}

__global__ void verify_pointers_kernel(hipComplex* const* A, hipComplex* const* B, hipComplex **C, const int batchCount)
{
    //print first element value
    //print values of A, B, C
    for(int i = 0; i < batchCount; i++)
    {
        printf("GEMMBatchSolver::verify_pointers_kernel: A[%d]: %p, B[%d]: %p, C[%d]: %p\n",
            i, (void*)A[i],i, (void*)B[i],i, (void*)C[i]);
        printf("GEMMBatchSolver::verify_pointers_kernel: A[%d][0]: (%f, %f), B[%d][0]: (%f, %f), C[%d][0]: (%f, %f)\n",
           i,hipCrealf(A[i][0]), hipCimagf(A[i][0]),
           i,hipCrealf(B[i][0]), hipCimagf(B[i][0]),
           i,hipCrealf(C[i][0]), hipCimagf(C[i][0]));
    }
}
void GEMMBatchSolver::gemm(hipComplex* const* A, hipComplex* const* B, hipComplex **C,
                      int M, int K, int N, int batchCount)
{
    const hipComplex alpha = make_hipComplex(1.0f, 0.0f);
    const hipComplex beta = make_hipComplex(0.0f, 0.0f);
    //print all array pointers
    //printf("GEMMSolver::gemm: A: %p, B: %p, C: %p\n", (void*)A, (void*)B, (void*)C);
    /*
    printf("GEMMBatchSolver::gemm: M: %d, K: %d, N: %d, batchCount: %d \n", M, K, N, batchCount);
    hipDeviceSynchronize(); // Ensure previous operations are completed before proceeding
    fflush(stdout); // Flush stdout to ensure logs are printed immediately
    verify_pointers_kernel<<<1, 1>>>(A, B, C, batchCount);
    hipDeviceSynchronize(); // Ensure previous operations are completed before proceeding
    fflush(stdout); // Flush stdout to ensure logs are printed immediately
*/
    // Perform batched GEMM operation
    hipblasCgemmBatched(m_handle,
                       HIPBLAS_OP_N, HIPBLAS_OP_N,
                       M, N, K,
                       &alpha,
                       A, M,
                       B, K,
                       &beta,
                       C, M,
                       batchCount);

}

