#include "hip/hip_runtime.h"
#include "GEMMSolver.h"
#include <hipblas.h>
#include <thrust/device_ptr.h>
#include <thrust/for_each.h>
#include <thrust/execution_policy.h>

GEMMSolver::GEMMSolver() {
    hipblasCreate(&m_handle);
}

GEMMSolver::~GEMMSolver() {
    hipblasDestroy(m_handle);
}

void GEMMSolver::gemm(const hipComplex* A, const hipComplex* B, hipComplex* C,
                      int M, int K, int N, hipStream_t stream) {
    const hipComplex alpha = make_hipComplex(1.0f, 0.0f);
    const hipComplex beta = make_hipComplex(0.0f, 0.0f);

    hipblasSetStream(m_handle, stream);
    hipblasCgemm(m_handle,
                HIPBLAS_OP_N, HIPBLAS_OP_N,
                M, N, K,
                &alpha,
                A, M,
                B, K,
                &beta,
                C, M);
}

void GEMMSolver::gather(const int* d_gather_map, hipComplex* d_out,
                        int num_rows, int num_cols, int matrixSize, hipStream_t stream) {
    size_t total = static_cast<size_t>(num_rows) * num_cols;

    for (int j = 0; j < num_cols; ++j) {
        thrust::gather(thrust::cuda::par.on(stream),
                       d_gather_map,
                       d_gather_map + num_rows,
                       thrust::device_pointer_cast(d_out + matrixSize * j),
                       thrust::device_pointer_cast(d_out + num_rows * j));
    }
}

__global__ void scatter_add_kernel(const int* map, hipComplex* global, const hipComplex* local,
                                   int num_rows, int num_cols, int matrixSize) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y;

    if (row < num_rows && col < num_cols) {
        int global_idx = map[row] + matrixSize * col;
        int local_idx = row + num_rows * col;

        atomicAdd(&global[global_idx].x, local[local_idx].x);
        atomicAdd(&global[global_idx].y, local[local_idx].y);
    }
}

void GEMMSolver::scatterAdd(const int* d_scatter_map, const hipComplex* d_in,
                            int num_rows, int num_cols, int matrixSize, hipStream_t stream) {
    int blockSize = 256;
    dim3 block(blockSize);
    dim3 grid((num_rows + blockSize - 1) / blockSize, num_cols);

    scatter_add_kernel<<<grid, block, 0, stream>>>(
        d_scatter_map, d_in /* global output */, d_in, num_rows, num_cols, matrixSize);
}
