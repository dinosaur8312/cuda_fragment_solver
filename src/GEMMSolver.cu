#include "hip/hip_runtime.h"
#include "GEMMSolver.h"
#include <hipblas.h>
#include <thrust/device_ptr.h>
#include <thrust/gather.h>
#include <thrust/for_each.h>
#include <thrust/execution_policy.h>

GEMMSolver::GEMMSolver()
    : m_handle(nullptr)
{
    hipblasCreate(&m_handle);
}

GEMMSolver::~GEMMSolver()
{
    std::cout << "Destroying GEMMSolver..." << std::endl;
  //  hipStreamSynchronize(m_stream); // Ensure all operations on the stream are completed before cleanup
    hipblasDestroy(m_handle);
}

void GEMMSolver::setStream(hipStream_t stream)
{
    m_stream = stream;
    hipblasSetStream(m_handle, m_stream);
}

__global__ void verify_pointers_kernel(const hipComplex *A, const hipComplex *B, hipComplex *C)
{
    //print first element value
    printf("GEMMSolver::verify_pointers_kernel: A: %p, B: %p, C: %p\n", (void*)A, (void*)B, (void*)C);
    //print values of A, B, C
    printf("GEMMSolver::verify_pointers_kernel: A: %f + %fi, B: %f + %fi, C: %f + %fi\n", A[0].x, A[0].y, B[0].x, B[0].y, C[0].x, C[0].y);
}

void GEMMSolver::gemm(const hipComplex *A, const hipComplex *B, hipComplex *C,
                      int M, int K, int N)
{
    const hipComplex alpha = make_hipComplex(1.0f, 0.0f);
    const hipComplex beta = make_hipComplex(0.0f, 0.0f);
    //print all array pointers
    //printf("GEMMSolver::gemm: A: %p, B: %p, C: %p\n", (void*)A, (void*)B, (void*)C);
    printf("GEMMSolver::gemm: M: %d, K: %d, N: %d\n", M, K, N);


    hipblasCgemm(m_handle,
                HIPBLAS_OP_N, HIPBLAS_OP_N,
                M, N, K,
                &alpha,
                A, M,
                B, K,
                &beta,
                C, M);

}

__global__ void gather_kernel(const int *d_gather_map,
                              const hipComplex *d_in,
                              hipComplex *d_out,
                              const int ldin,
                              const int ldout)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int rep = blockIdx.y;

    if (idx < ldin)
    {
        int gather_idx = d_gather_map[idx];
        d_out[gather_idx + rep * ldout] = d_in[idx + rep * ldin];
    }
}

void GEMMSolver::gather(const int *d_gather_map,
                        const hipComplex *d_in,
                        hipComplex *d_out,
                        const int ldin,
                        const int ldout,
                        const int reps)
{
    dim3 block(256);
    dim3 grid(iDivUp(ldin, block.x), reps);
    gather_kernel<<<grid, block, 0, m_stream>>>(d_gather_map, d_in, d_out, ldin, ldout);
}

__global__ void scatter_kernel(const int *d_scatter_map,
                               const hipComplex *d_in,
                               hipComplex *d_out,
                               const int ldin,
                               const int ldout)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int rep = blockIdx.y;

    if (idx < ldin)
    {
        int scatter_idx = d_scatter_map[idx];
        d_out[scatter_idx + rep * ldout] += d_in[idx + rep * ldin];
    }
}

void GEMMSolver::scatter(const int *d_scatter_map, 
                         const hipComplex *d_in,
                         hipComplex *d_out,
                         const int ldin,
                         const int ldout,
                         const int reps)
{
    int blockSize = 256;
    dim3 block(blockSize);
    dim3 grid(iDivUp(ldin, blockSize), reps);

    scatter_kernel<<<grid, block, 0, m_stream>>>(
        d_scatter_map, d_in, d_out, ldin, ldout);
}
