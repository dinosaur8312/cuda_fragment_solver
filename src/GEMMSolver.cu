#include "hip/hip_runtime.h"
#include "GEMMSolver.h"
#include <hipblas.h>
#include <thrust/device_ptr.h>
#include <thrust/gather.h>
#include <thrust/for_each.h>
#include <thrust/execution_policy.h>

GEMMSolver::GEMMSolver(hipStream_t stream)
    : m_stream(stream), m_handle(nullptr)
{
    hipblasCreate(&m_handle);
    hipblasSetStream(m_handle, m_stream);
}

GEMMSolver::~GEMMSolver()
{
    std::cout << "Destroying GEMMSolver..." << std::endl;
    hipStreamSynchronize(m_stream); // Ensure all operations on the stream are completed before cleanup
    hipblasDestroy(m_handle);
}

void GEMMSolver::gemm(const hipComplex *A, const hipComplex *B, hipComplex *C,
                      int M, int K, int N)
{
    const hipComplex alpha = make_hipComplex(1.0f, 0.0f);
    const hipComplex beta = make_hipComplex(0.0f, 0.0f);

    hipblasCgemm(m_handle,
                HIPBLAS_OP_N, HIPBLAS_OP_N,
                M, N, K,
                &alpha,
                A, M,
                B, K,
                &beta,
                C, M);
}

__global__ void gather_kernel(const int *d_gather_map,
                              const hipComplex *d_in,
                              hipComplex *d_out,
                              const int ldin,
                              const int ldout)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int rep = blockIdx.y;

    if (idx < ldin)
    {
        int gather_idx = d_gather_map[idx];
        d_out[gather_idx + rep * ldout] = d_in[idx + rep * ldin];
    }
}

void GEMMSolver::gather(const int *d_gather_map,
                        const hipComplex *d_in,
                        hipComplex *d_out,
                        const int ldin,
                        const int ldout,
                        const int reps)
{
    dim3 block(256);
    dim3 grid(iDivUp(ldin, block.x), reps);
    gather_kernel<<<grid, block, 0, m_stream>>>(d_gather_map, d_in, d_out, ldin, ldout);
}

__global__ void scatter_kernel(const int *d_scatter_map,
                               const hipComplex *d_in,
                               hipComplex *d_out,
                               const int ldin,
                               const int ldout)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int rep = blockIdx.y;

    if (idx < ldin)
    {
        int scatter_idx = d_scatter_map[idx];
        d_out[scatter_idx + rep * ldout] += d_in[idx + rep * ldin];
    }
}

void GEMMSolver::scatter(const int *d_scatter_map, 
                         const hipComplex *d_in,
                         hipComplex *d_out,
                         const int ldin,
                         const int ldout,
                         const int reps)
{
    int blockSize = 256;
    dim3 block(blockSize);
    dim3 grid(iDivUp(ldin, blockSize), reps);

    scatter_kernel<<<grid, block, 0, m_stream>>>(
        d_scatter_map, d_in, d_out, ldin, ldout);
}
