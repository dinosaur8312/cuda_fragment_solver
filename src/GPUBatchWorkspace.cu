#include "GPUBatchWorkspace.h"
#include <hip/hip_runtime_api.h>
#include <cassert>

GPUBatchWorkspace::GPUBatchWorkspace(int nRHS, int matrixSize)
    : m_stream(nullptr), m_nRHS(nRHS), m_matrixSize(matrixSize)
{
    allocateGlobalMats();
}

GPUBatchWorkspace::~GPUBatchWorkspace()
{
    releaseLocalMats();
    releaseGlobalMats();
}

void GPUBatchWorkspace::ensureSize(int M, int N, int R, int tasknum)
{
    // Implement logic to ensure local buffer sizes for batch tasks
    // (similar to GPUWorkspace, but for batched operations)
    // ...
    //hipDeviceSynchronize(); // Ensure previous operations are completed before resizing
    //fflush(stdout); // Flush stdout to ensure logs are printed before resizing
    //printf("[GPUBatchWorkspace] Ensuring size for tasknum %d: M=%d, N=%d, R=%d\n", tasknum, M, N, R);
    size_t need_LocalBSize = N * m_nRHS * sizeof(hipComplex) * tasknum;
    size_t need_LocalCSize = M * m_nRHS * sizeof(hipComplex) * tasknum;
    size_t need_localMatSize = R * m_nRHS * sizeof(hipComplex) * tasknum;
    size_t need_denseMatSize = M * N * sizeof(hipComplex) * tasknum;
    size_t need_QmatSize = M * R * sizeof(hipComplex) * tasknum;
    size_t need_RmatSize = R * N * sizeof(hipComplex) * tasknum;

    bool taskSizeChanged = false;
    bool LocalBSizeChanged = false;
    bool LocalCSizeChanged = false;
    bool localMatSizeChanged = false;
    bool denseMatSizeChanged = false;
    bool QmatSizeChanged = false;
    bool RmatSizeChanged = false;
    if (tasknum > m_tasknum)
    {
        // If the number of tasks increases, we need to allocate more memory
        m_tasknum = tasknum;
        taskSizeChanged = true;
        //printf("[GPUBatchWorkspace] Task number increased to %d\n", m_tasknum);
        if (d_denseMat_array)
        {
            hipFreeAsync(d_denseMat_array, m_stream);
            d_denseMat_array = nullptr;
        }
        hipMallocAsync(&d_denseMat_array, tasknum * sizeof(hipComplex *), m_stream);
        if (d_Qmat_array)
        {
            hipFreeAsync(d_Qmat_array, m_stream);
            d_Qmat_array = nullptr;
        }
        hipMallocAsync(&d_Qmat_array, tasknum * sizeof(hipComplex *), m_stream);
        if (d_Rmat_array)
        {
            hipFreeAsync(d_Rmat_array, m_stream);
            d_Rmat_array = nullptr;
        }
        hipMallocAsync(&d_Rmat_array, tasknum * sizeof(hipComplex *), m_stream);

        if (d_localB_array)
        {
            hipFreeAsync(d_localB_array, m_stream);
            d_localB_array = nullptr;
        }
        hipMallocAsync(&d_localB_array, tasknum * sizeof(hipComplex *), m_stream);
        if (d_localC_array)
        {
            hipFreeAsync(d_localC_array, m_stream);
            d_localC_array = nullptr;
        }
        hipMallocAsync(&d_localC_array, tasknum * sizeof(hipComplex *), m_stream);
        if (d_localMat_array)
        {
            hipFreeAsync(d_localMat_array, m_stream);
            d_localMat_array = nullptr;
        }
        hipMallocAsync(&d_localMat_array, tasknum * sizeof(hipComplex *), m_stream);
    }

    if (need_LocalBSize > m_localBSize)
    {
        LocalBSizeChanged = true;
        //printf("[GPUBatchWorkspace] LocalB size increased to %zu\n", need_LocalBSize);
        if (d_localB_)
            hipFreeAsync(d_localB_, m_stream);
        hipMallocAsync(&d_localB_, need_LocalBSize, m_stream);


        if (h_tempB_)
            hipHostFree(h_tempB_);
        hipHostMalloc(&h_tempB_, need_LocalBSize);
        m_localBSize = need_LocalBSize;
    }

    {
        std::vector<hipComplex *> localB_array(tasknum, nullptr);
        for (int i = 0; i < tasknum; ++i)
        {
            localB_array[i] = d_localB_ + i * N * m_nRHS;
        }
        hipMemcpyAsync(d_localB_array, localB_array.data(), tasknum * sizeof(hipComplex *), hipMemcpyHostToDevice, m_stream);
    }

    if (need_LocalCSize > m_localCSize)
    {
        LocalCSizeChanged = true;
        if (d_localC_)
            hipFreeAsync(d_localC_, m_stream);
        hipMallocAsync(&d_localC_, need_LocalCSize, m_stream);

        if (h_tempC_)
            hipHostFree(h_tempC_);
        hipHostMalloc(&h_tempC_, need_LocalCSize);
        m_localCSize = need_LocalCSize;
    }
    {
        std::vector<hipComplex *> localC_array(tasknum, nullptr);
        for (int i = 0; i < tasknum; ++i)
        {
            localC_array[i] = d_localC_ + i * M * m_nRHS;
        }
        hipMemcpyAsync(d_localC_array, localC_array.data(), tasknum * sizeof(hipComplex *), hipMemcpyHostToDevice, m_stream);
    }

    if (need_localMatSize > m_localMatSize)
    {
        localMatSizeChanged = true;
        if (d_localMat_)
            hipFreeAsync(d_localMat_, m_stream);
        hipMallocAsync(&d_localMat_, need_localMatSize, m_stream);

        m_localMatSize = need_localMatSize;
    }
    //if((localMatSizeChanged)||(taskSizeChanged))
    {
        std::vector<hipComplex *> localMat_array(tasknum, nullptr);
        for (int i = 0; i < tasknum; ++i)
        {
            localMat_array[i] = d_localMat_ + i * R * m_nRHS;
        }
        hipMemcpyAsync(d_localMat_array, localMat_array.data(), tasknum * sizeof(hipComplex *), hipMemcpyHostToDevice, m_stream);
    }

    if (R == 0)
    {
        if (need_denseMatSize > m_denseMatSize)
        {
            denseMatSizeChanged = true;
            if (d_denseMat_)
                hipFreeAsync(d_denseMat_, m_stream);
            hipMallocAsync(&d_denseMat_, need_denseMatSize, m_stream);

            m_denseMatSize = need_denseMatSize;
        }
       // if(denseMatSizeChanged||taskSizeChanged)
        {
            std::vector<hipComplex *> denseMat_array(tasknum, nullptr);
            for (int i = 0; i < tasknum; ++i)
            {
                denseMat_array[i] = d_denseMat_ + i * M * N;
            }
            hipMemcpyAsync(d_denseMat_array, denseMat_array.data(), tasknum * sizeof(hipComplex *), hipMemcpyHostToDevice, m_stream);
        }
    }
    else
    {
        if (need_QmatSize > m_QmatSize)
        {
            QmatSizeChanged = true;
            if (d_Qmat_)
                hipFreeAsync(d_Qmat_, m_stream);
            hipMallocAsync(&d_Qmat_, need_QmatSize, m_stream);

            m_QmatSize = need_QmatSize;
        }
       // if (QmatSizeChanged || taskSizeChanged)
        {
            std::vector<hipComplex *> Qmat_array(tasknum, nullptr);
            for (int i = 0; i < tasknum; ++i)
            {
                Qmat_array[i] = d_Qmat_ + i * M * R;
            }
            hipMemcpyAsync(d_Qmat_array, Qmat_array.data(), tasknum * sizeof(hipComplex *), hipMemcpyHostToDevice, m_stream);
        }
        

        if (need_RmatSize > m_RmatSize)
        {
            RmatSizeChanged = true;
            if (d_Rmat_)
                hipFreeAsync(d_Rmat_, m_stream);
            hipMallocAsync(&d_Rmat_, need_RmatSize, m_stream);

            m_RmatSize = need_RmatSize;
        }
       // if(RmatSizeChanged||taskSizeChanged)
        {
            std::vector<hipComplex *> Rmat_array(tasknum, nullptr);
            for (int i = 0; i < tasknum; ++i)
            {
                Rmat_array[i] = d_Rmat_ + i * R * N;
            }
            hipMemcpyAsync(d_Rmat_array, Rmat_array.data(), tasknum * sizeof(hipComplex *), hipMemcpyHostToDevice, m_stream);
        }
    }

   // fflush(stdout); // Ensure logs are printed immediately
    return;
}

size_t GPUBatchWorkspace::getAvailableGPUMemory() const
{
    size_t free, total;
    hipMemGetInfo(&free, &total);
    return free;
}

void GPUBatchWorkspace::printMemoryInfo() const
{
    size_t free, total;
    hipMemGetInfo(&free, &total);
    printf("[GPUBatchWorkspace] Free GPU memory: %zu / %zu bytes\n", free, total);
}

void GPUBatchWorkspace::releaseLocalMats()
{
    if (d_localB_)
        hipFreeAsync(d_localB_, m_stream);
    if (d_localC_)
        hipFreeAsync(d_localC_, m_stream);
    if (d_localMat_)
        hipFreeAsync(d_localMat_, m_stream);
    d_localB_ = d_localC_ = d_localMat_ = nullptr;
    if (h_tempB_)
        hipHostFree(h_tempB_);
    if (h_tempC_)
        hipHostFree(h_tempC_);
    h_tempB_ = h_tempC_ = nullptr;

    if (d_denseMat_)
        hipFreeAsync(d_denseMat_, m_stream);
    if (d_Qmat_)
        hipFreeAsync(d_Qmat_, m_stream);
    if (d_Rmat_)
        hipFreeAsync(d_Rmat_, m_stream);
    d_denseMat_ = d_Qmat_ = d_Rmat_ = nullptr;

    if (d_denseMat_array)
    {
        hipFreeAsync(d_denseMat_array, m_stream);
        d_denseMat_array = nullptr;
    }
    if (d_Qmat_array)
    {
        hipFreeAsync(d_Qmat_array, m_stream);
        d_Qmat_array = nullptr;
    }
    if (d_Rmat_array)
    {
        hipFreeAsync(d_Rmat_array, m_stream);
        d_Rmat_array = nullptr;
    }
    if (d_localB_array)
    {
        hipFreeAsync(d_localB_array, m_stream);
        d_localB_array = nullptr;
    }
    if (d_localC_array)
    {
        hipFreeAsync(d_localC_array, m_stream);
        d_localC_array = nullptr;
    }
    if (d_localMat_array)
    {
        hipFreeAsync(d_localMat_array, m_stream);
        d_localMat_array = nullptr;
    }
    m_denseMatSize = 0;
    m_localMatSize = 0;
    m_localBSize = 0;
    m_localCSize = 0;
    m_QmatSize = 0;
    m_RmatSize = 0;
    m_tasknum = 0; // Reset task number
}

void GPUBatchWorkspace::allocateGlobalMats()
{
    if (!h_globalMatC_)
    {
        size_t sizeC = m_matrixSize * m_nRHS * sizeof(hipComplex);
        h_globalMatC_ = (hipComplex *)malloc(sizeC);
    }
}

void GPUBatchWorkspace::releaseGlobalMats()
{
    if (h_globalMatC_)
    {
        free(h_globalMatC_);
        h_globalMatC_ = nullptr;
    }
}
